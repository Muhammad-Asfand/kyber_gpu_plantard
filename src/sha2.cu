
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>

#define blocks crypto_hashblocks_sha256

__constant__ static const uint8_t iv[32] = {
  0x6a,0x09,0xe6,0x67,
  0xbb,0x67,0xae,0x85,
  0x3c,0x6e,0xf3,0x72,
  0xa5,0x4f,0xf5,0x3a,
  0x51,0x0e,0x52,0x7f,
  0x9b,0x05,0x68,0x8c,
  0x1f,0x83,0xd9,0xab,
  0x5b,0xe0,0xcd,0x19,
} ;

__device__ static uint32_t load_bigendian(const uint8_t *x)
{
  return
      (uint32_t) (x[3]) \
  | (((uint32_t) (x[2])) << 8) \
  | (((uint32_t) (x[1])) << 16) \
  | (((uint32_t) (x[0])) << 24)
  ;
}

__device__ static void store_bigendian(uint8_t *x,uint32_t u)
{
  x[3] = u; u >>= 8;
  x[2] = u; u >>= 8;
  x[1] = u; u >>= 8;
  x[0] = u;
}

#define SHR(x,c) ((x) >> (c))
#define ROTR(x,c) (((x) >> (c)) | ((x) << (32 - (c))))

#define Ch(x,y,z) ((x & y) ^ (~x & z))
#define Maj(x,y,z) ((x & y) ^ (x & z) ^ (y & z))
#define Sigma0(x) (ROTR(x, 2) ^ ROTR(x,13) ^ ROTR(x,22))
#define Sigma1(x) (ROTR(x, 6) ^ ROTR(x,11) ^ ROTR(x,25))
#define sigma0(x) (ROTR(x, 7) ^ ROTR(x,18) ^ SHR(x, 3))
#define sigma1(x) (ROTR(x,17) ^ ROTR(x,19) ^ SHR(x,10))

#define M(w0,w14,w9,w1) w0 = sigma1(w14) + w9 + sigma0(w1) + w0;

#define EXPAND \
  M(w0 ,w14,w9 ,w1 ) \
  M(w1 ,w15,w10,w2 ) \
  M(w2 ,w0 ,w11,w3 ) \
  M(w3 ,w1 ,w12,w4 ) \
  M(w4 ,w2 ,w13,w5 ) \
  M(w5 ,w3 ,w14,w6 ) \
  M(w6 ,w4 ,w15,w7 ) \
  M(w7 ,w5 ,w0 ,w8 ) \
  M(w8 ,w6 ,w1 ,w9 ) \
  M(w9 ,w7 ,w2 ,w10) \
  M(w10,w8 ,w3 ,w11) \
  M(w11,w9 ,w4 ,w12) \
  M(w12,w10,w5 ,w13) \
  M(w13,w11,w6 ,w14) \
  M(w14,w12,w7 ,w15) \
  M(w15,w13,w8 ,w0 )

#define F(w,k) \
  T1 = h + Sigma1(e) + Ch(e,f,g) + k + w; \
  T2 = Sigma0(a) + Maj(a,b,c); \
  h = g; \
  g = f; \
  f = e; \
  e = d + T1; \
  d = c; \
  c = b; \
  b = a; \
  a = T1 + T2;

__device__ static int crypto_hashblocks_sha256(uint8_t *statebytes,const uint8_t *in,size_t inlen)
{
  uint32_t state[8];
  uint32_t a;
  uint32_t b;
  uint32_t c;
  uint32_t d;
  uint32_t e;
  uint32_t f;
  uint32_t g;
  uint32_t h;
  uint32_t T1;
  uint32_t T2;

  a = load_bigendian(statebytes +  0); state[0] = a;
  b = load_bigendian(statebytes +  4); state[1] = b;
  c = load_bigendian(statebytes +  8); state[2] = c;
  d = load_bigendian(statebytes + 12); state[3] = d;
  e = load_bigendian(statebytes + 16); state[4] = e;
  f = load_bigendian(statebytes + 20); state[5] = f;
  g = load_bigendian(statebytes + 24); state[6] = g;
  h = load_bigendian(statebytes + 28); state[7] = h;

  while (inlen >= 64) {
    uint32_t w0  = load_bigendian(in +  0);
    uint32_t w1  = load_bigendian(in +  4);
    uint32_t w2  = load_bigendian(in +  8);
    uint32_t w3  = load_bigendian(in + 12);
    uint32_t w4  = load_bigendian(in + 16);
    uint32_t w5  = load_bigendian(in + 20);
    uint32_t w6  = load_bigendian(in + 24);
    uint32_t w7  = load_bigendian(in + 28);
    uint32_t w8  = load_bigendian(in + 32);
    uint32_t w9  = load_bigendian(in + 36);
    uint32_t w10 = load_bigendian(in + 40);
    uint32_t w11 = load_bigendian(in + 44);
    uint32_t w12 = load_bigendian(in + 48);
    uint32_t w13 = load_bigendian(in + 52);
    uint32_t w14 = load_bigendian(in + 56);
    uint32_t w15 = load_bigendian(in + 60);

    F(w0 ,0x428a2f98)
    F(w1 ,0x71374491)
    F(w2 ,0xb5c0fbcf)
    F(w3 ,0xe9b5dba5)
    F(w4 ,0x3956c25b)
    F(w5 ,0x59f111f1)
    F(w6 ,0x923f82a4)
    F(w7 ,0xab1c5ed5)
    F(w8 ,0xd807aa98)
    F(w9 ,0x12835b01)
    F(w10,0x243185be)
    F(w11,0x550c7dc3)
    F(w12,0x72be5d74)
    F(w13,0x80deb1fe)
    F(w14,0x9bdc06a7)
    F(w15,0xc19bf174)

    EXPAND

    F(w0 ,0xe49b69c1)
    F(w1 ,0xefbe4786)
    F(w2 ,0x0fc19dc6)
    F(w3 ,0x240ca1cc)
    F(w4 ,0x2de92c6f)
    F(w5 ,0x4a7484aa)
    F(w6 ,0x5cb0a9dc)
    F(w7 ,0x76f988da)
    F(w8 ,0x983e5152)
    F(w9 ,0xa831c66d)
    F(w10,0xb00327c8)
    F(w11,0xbf597fc7)
    F(w12,0xc6e00bf3)
    F(w13,0xd5a79147)
    F(w14,0x06ca6351)
    F(w15,0x14292967)

    EXPAND

    F(w0 ,0x27b70a85)
    F(w1 ,0x2e1b2138)
    F(w2 ,0x4d2c6dfc)
    F(w3 ,0x53380d13)
    F(w4 ,0x650a7354)
    F(w5 ,0x766a0abb)
    F(w6 ,0x81c2c92e)
    F(w7 ,0x92722c85)
    F(w8 ,0xa2bfe8a1)
    F(w9 ,0xa81a664b)
    F(w10,0xc24b8b70)
    F(w11,0xc76c51a3)
    F(w12,0xd192e819)
    F(w13,0xd6990624)
    F(w14,0xf40e3585)
    F(w15,0x106aa070)

    EXPAND

    F(w0 ,0x19a4c116)
    F(w1 ,0x1e376c08)
    F(w2 ,0x2748774c)
    F(w3 ,0x34b0bcb5)
    F(w4 ,0x391c0cb3)
    F(w5 ,0x4ed8aa4a)
    F(w6 ,0x5b9cca4f)
    F(w7 ,0x682e6ff3)
    F(w8 ,0x748f82ee)
    F(w9 ,0x78a5636f)
    F(w10,0x84c87814)
    F(w11,0x8cc70208)
    F(w12,0x90befffa)
    F(w13,0xa4506ceb)
    F(w14,0xbef9a3f7)
    F(w15,0xc67178f2)

    a += state[0];
    b += state[1];
    c += state[2];
    d += state[3];
    e += state[4];
    f += state[5];
    g += state[6];
    h += state[7];

    state[0] = a;
    state[1] = b;
    state[2] = c;
    state[3] = d;
    state[4] = e;
    state[5] = f;
    state[6] = g;
    state[7] = h;

    in += 64;
    inlen -= 64;
  }

  store_bigendian(statebytes +  0,state[0]);
  store_bigendian(statebytes +  4,state[1]);
  store_bigendian(statebytes +  8,state[2]);
  store_bigendian(statebytes + 12,state[3]);
  store_bigendian(statebytes + 16,state[4]);
  store_bigendian(statebytes + 20,state[5]);
  store_bigendian(statebytes + 24,state[6]);
  store_bigendian(statebytes + 28,state[7]);

  return inlen;
}

__global__ void sha256_gpu(uint8_t *in, uint8_t *out, uint32_t inlen)
{
  uint8_t h[32];  
  uint8_t padded[128];
  unsigned int i;

  size_t bits = inlen << 3;
  for (i = 0;i < 32;++i) h[i] = iv[i];

  blocks(h,in,inlen);
  in += inlen;
  inlen &= 63;
  in -= inlen;

  for (i = 0;i < inlen;++i) padded[i] = in[i];
  padded[inlen] = 0x80;

  if (inlen < 56) {
    for (i = inlen + 1;i < 56;++i) padded[i] = 0;
    padded[56] = bits >> 56;
    padded[57] = bits >> 48;
    padded[58] = bits >> 40;
    padded[59] = bits >> 32;
    padded[60] = bits >> 24;
    padded[61] = bits >> 16;
    padded[62] = bits >> 8;
    padded[63] = bits;
    blocks(h,padded,64);
  } else {
    for (i = inlen + 1;i < 120;++i) padded[i] = 0;
    padded[120] = bits >> 56;
    padded[121] = bits >> 48;
    padded[122] = bits >> 40;
    padded[123] = bits >> 32;
    padded[124] = bits >> 24;
    padded[125] = bits >> 16;
    padded[126] = bits >> 8;
    padded[127] = bits;
    blocks(h,padded,128);
  }

  for (i = 0;i < 32;++i) out[i] = h[i];
  // for (i = 0;i < 32;++i) {if(i%4==0) printf("\n");printf("%x ", out[i]); }
}